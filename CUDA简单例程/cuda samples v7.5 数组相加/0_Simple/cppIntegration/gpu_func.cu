#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/*
* ��������ƴͼ�����GPU�ⷨ
*/

// System includes
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

/*
������  ��addOne
�������ܣ�����+1����
���������
num      :��Ҫ��1����
*/
__device__ void addOne(int &num)
{
	num++;
}
/*
������  ��gpuTestSiukwanAddOne
�������ܣ�����a_gpu[i] * b_gpu[i]�����ѽ����ŵ�c_gpu[i]�У�ʵ���豸�˵ĺ�������
���������
a_gpu    :�����ľ���
b_gpu    :��������ͼ��
c_gpu    :��������ĵ�һ���ո�λ��x
*/
__global__ void gpuTestSiukwanAddOne(int *const a_gpu, int *const b_gpu, int *const c_gpu)
{
	// write data to global memory
	const unsigned int tid = (blockIdx.x*blockDim.x) + threadIdx.x;
	c_gpu[tid] = a_gpu[tid] * b_gpu[tid];
	for (int i = 0; i < 10240; i++)
		addOne(c_gpu[tid]);
}

/*
������  ��gpuTestSiukwan
�������ܣ�����a_gpu[i] * b_gpu[i]�����ѽ����ŵ�c_gpu[i]��,
���������
a_gpu    :�����ľ���
b_gpu    :��������ͼ��
c_gpu    :��������ĵ�һ���ո�λ��x
*/
__global__ void gpuTestSiukwan(int *const a_gpu, int *const b_gpu, int *const c_gpu)
{
	// write data to global memory
	const unsigned int tid = (blockIdx.x*blockDim.x) + threadIdx.x;
	c_gpu[tid] = a_gpu[tid] * b_gpu[tid];
	for (int i = 0; i < 10240; i++)
		c_gpu[tid]++;
}

//�ӿں���
extern "C" void runCUDA(int blocks, int threads, int *const a_gpu, int *const b_gpu, int *const c_gpu)
{
	gpuTestSiukwan << < blocks, threads >> >(a_gpu, b_gpu, c_gpu);
}

//�ӿں���
extern "C" void runCUDA_AddOne(int blocks, int threads, int *const a_gpu, int *const b_gpu, int *const c_gpu)
{
	gpuTestSiukwanAddOne << < blocks, threads >> >(a_gpu, b_gpu, c_gpu);
}