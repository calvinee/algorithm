#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/*
* ��������ƴͼ�����GPU�ⷨ
*/

// System includes
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#ifndef MAX
#define MAX(a,b) (a > b ? a : b)
#endif

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
////////////////////////////////////////////////////////////////////////////////
/*��¼����ͼ�εĳ������״*/
struct GraphNode
{
	char shape[5][5];      //ͼ�ε���״����ΪboolҲռ����1B������ʹ��char����bool������
	int x;                //ͼ�εĸ߶�
	int y;                 //ͼ�εĿ��
	int fill_x;            //ͼ�εĵ�һ����ֵ�������x
	int fill_y;            //ͼ�εĵ�һ����ֵ�������y
};

/*
*��¼ÿ��ͼ�εı��Σ�
*����ԭʼ����ת����ת��
*�����8��
*/
struct GraphFormat
{
	GraphNode format[8];   //ͼ�εı��Σ������8��
	int formatCount;       //ͼ�εı�������
	char c;                //ͼ�εı�ţ�123456789abc
};

/*
*�洢����ͼ��
*һ��12��ͼ��
*/
struct GraphAll
{
	GraphFormat graph[12]; //����ͼ��
	int graphCount;        //ͼ������
};

/*
*��Ҫ���ľ������ݽṹ
*ÿ��GPU�̷߳���һ��
*/
struct MatrixNode{
	char shape[20][20];    //ȷ����12����ʼͼ�Σ�ͼ�δ���3*3��С�ģ�����С��3�ģ������ڴ𰸡����ٴ�3*20��ʼ��������״��Сֱ�Ӷ���Ϊ20*20
	int x, y;              //����ĳ���
	bool solution[12][8];  //�������һ�������Ŀ�����
	bool graphUsed[12];    //��¼12��ͼ�ε�ʹ�����
	int thisLevelCount;    //�Ե�ǰ����Ϊ�������ܹ������ͼ�ε�����
	int depth;             //�����ȣ�Ҳ�൱���Ѿ�����˶��ٸ�ͼ��
};


///////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_graph  memory to process (in and out)
///////////////////////////////////////////////////////////////////////////////
__global__ void gpuTestSiukwan(GraphAll *const g_graph)
{
	// write data to global memory
	const unsigned int tid = threadIdx.x;
	for (int k = 0; k < g_graph->graph[tid].formatCount; k++)
	{
		for (int i = 0; i < g_graph->graph[tid].format[k].x; i++)
		{
			for (int j = 0; j < g_graph->graph[tid].format[k].y; j++)
			{
				//g_graph->graph[tid].format[k].shape[i][j] += '0';
			}
		}
	}
}

extern "C" bool
SiukwanTest(GraphAll *const g_graph)
{
	gpuTestSiukwan << < 1, 12 >> >(g_graph);
	return true;
}




/*
������  ��getFirstUnfill
�������ܣ�������mat�е�һ���հ׵ĸ��ӣ����꣩
���������
mat:����mat
*/
__device__ void getFirstUnfill(MatrixNode &g_mat, int &x, int&y)
{
	for (int i = 0; i <g_mat.x; i++)
	{
		for (int j = 0; j < g_mat.y; j++)
		{//�ҳ���һ��δ���λ�ã���¼������ֱ������
			if (g_mat.shape[i][j] == 0)
			{//Ϊ0����û�����
				x = i;
				y = j;
				break;//�ҵ���һ������������
			}
		}
		if (x != -1) break;
	}
}
/*
������  ��getFirstFill
�������ܣ����graph�е�һ����ֵ��λ�ã����꣩
���������
graph:����graph
*/
__device__ void getFirstFill(GraphNode &graph, int &x, int&y)
{
	for (int j = 0; j < graph.x; j++)
	{
		for (int h = 0; h < graph.y; h++)
		{//�ҳ���һ��δ���λ�ã���¼������ֱ������
			if (graph.shape[j][h] != 0)
			{//��Ϊ0������ֵ
				x = j;
				y = h;
				break;
			}
		}
		if (x != -1) break;
	}
}

/*
������  ��canFillMatrix
�������ܣ�ͼ���ܷ���䵽������
���������
g_mat    :�����ľ���
graph    :��������ͼ��
toFill_x :��������ĵ�һ���ո�λ��x
toFill_y :��������ĵ�һ���ո�λ��y
fill_x   :����ͼ�εĵ�һ���ǿո�λ��x
fill_y   :����ͼ�εĵ�һ���ǿո�λ��y
*/
__device__ bool canFillMatrix(MatrixNode &g_mat, GraphNode &graph, int&toFill_x, int&toFill_y,  int&fill_x, int&fill_y)
{//�ж�ͼ���ܷ���䵽�����У����о���ĵ�һ���հ׵�����ΪtoFill


	for (int i = 0; i < graph.x; i++)
	{
		for (int j = 0; j < graph.y; j++)
		{
			if (i + toFill_x - fill_x >= g_mat.x ||
				j + toFill_y - fill_y >= g_mat.y ||
				i + toFill_x - fill_x < 0 ||
				j + toFill_y - fill_y < 0
				)
			{//ͼ������Խ�磬����false
				return false;
			}
			else if (graph.shape[i][j] != 0 && g_mat.shape[i + toFill_x - fill_x][j + toFill_y - fill_y] != 0)
			{//ͼ�����Ĳ��ֲ�Ϊ�գ�����false
				return false;
			}
		}
	}
	return true;
}



__global__ void dfsCUDA_LastVersion(MatrixNode *const g_mat, GraphAll *const g_graph, int*solutionSum)
{
	// write data to global memory
	const unsigned int tid = (blockIdx.x*blockDim.x) + threadIdx.x;
	if (tid >= solutionSum[0]) return;

	//��ȡδ���ĵ�һ���ո�
	int unfill_x = -1, unfill_y = -1;
	getFirstUnfill(g_mat[tid], unfill_x, unfill_y);

	//һ��ʼֻ��12���̣߳��߳�id��ͼ��id
	if (g_mat[tid].depth == 0)
	{
		for (int j = 0; j < g_graph->graph[tid].formatCount; j++)
		{
			if (canFillMatrix(g_mat[tid], g_graph->graph[tid].format[j], unfill_x, unfill_y, g_graph->graph[tid].format[j].fill_x, g_graph->graph[tid].format[j].fill_y))
			{//��¼i��j����¼��һ����Щ�𰸿���
				//�ò��������
				g_mat[tid].thisLevelCount++;
				g_mat[tid].solution[tid][j] = true;//��¼����i��j����
			}
		}
	}
	else
	{
		//�����������12��ͼ��
		for (int i = 0; i < 12; i++)
		{
			if (g_mat[tid].graphUsed[i])
				continue;
			for (int j = 0; j < g_graph->graph[i].formatCount; j++)
			{
				if (canFillMatrix(g_mat[tid], g_graph->graph[i].format[j], unfill_x, unfill_y, g_graph->graph[i].format[j].fill_x, g_graph->graph[i].format[j].fill_y))
				{//��¼i��j����¼��һ����Щ�𰸿���
					//�ò��������
					g_mat[tid].thisLevelCount++;
					g_mat[tid].solution[i][j] = true;//��¼����i��j����
				}
			}
		}
	}
	//����++
	g_mat[tid].depth++;
}

//�ӿں���
extern "C" void runCUDA(int blocks, int threads, MatrixNode *const g_mat, GraphAll *const g_graph, int *solutionSum)
{
	dfsCUDA_LastVersion << < blocks, threads >> >(g_mat, g_graph, solutionSum);
}

